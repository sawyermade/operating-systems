#include "hip/hip_runtime.h"
/* ==================================================================
	Programmer 1: Daniel Sawyer (danielsawyer@mail.usf.edu)
	Programmer 2: Hunter Morera (hmorera@mail.usf.edu)
	Programmer 3: Kevin Hicks
	Programmer 4: Conner Wulf
	Linkage Covariance Matrix with respect to privacy
	To compile: run make in directory, outputs pj-final binary
	Libs Needed: igraph and thrust
   ==================================================================
*/

//INCLUDES
#include <stdio.h>
#include <stdlib.h>
#include <igraph/igraph.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

//STRUCTS
typedef struct {
  int *array;
  size_t used;
  size_t size;
} Array;

//GLOBAL VARS
igraph_neimode_t OUTALL;

//NAIVE KERNELS & PREP
void Naive_Prep(igraph_t &graph);
__global__ void Naive(int* d_matrix, int* d_result, int n_vertices);
__global__ void Naive_Hist(int* d_result, int* d_hist, int n_vertices);

//OPTIMIZATION 1 KERNELS & PREP
void OPT_1_PREP(igraph_t &graph);
__global__ void OPT_1(int* adj, int* lcm, int* sizes, int n);
__global__ void OPT_1_HIST(int* lcm, int* hist, int n);

//OPTIMIZED 2 SATHYAS GPU 1
int OPT_2_PREP(igraph_t &graph, igraph_neimode_t OUTALL, int numThreads);
__global__ void OPT_2_SIZES(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices);
__global__ void OPT_2(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices);

//OPTIMIZATION 3 KERNELS & PREP
void OPT_3_PREP(igraph_t &graph);
__global__ void OPT_3_SIZES(int* adj, int* lcmsizes, int* sizes, int n);
__global__ void OPT_3_SIZES_SUM(int* lcmsizes, int n);
__global__ void OPT_3(int* adj, int* lcm, int* sizes, int* lcmsizes, int n);
__global__ void OPT_3_HIST(int* lcm, int* hist, int* lcmsizes, int n);

//OPTIMIZED 4 SATHYAS GPU 2
int OPT_4_PREP(igraph_t &graph, igraph_neimode_t OUTALL, int numThreads);
__global__ void OPT_4_SIZES(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices);
__global__ void OPT_4(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices);
__global__ void OPT_4_HIST(int *d_lcmMatrix, int *d_LCMSize, int *d_histogram, int n_vertices);

//OPTIMIZED CPU
void LCM_CPU_Kernel(long int **adjList, int *sizeAdj, int n_vertices);
void LCM_CPU(igraph_t &graph, igraph_neimode_t OUTALL);
void initArray(Array *a, size_t initialSize);
void insertArray(Array *a, int element);
void freeArray(Array *a);
int commonNeighbor(long int arr1[], long int arr2[], int m, int n);
int equalArray(Array a1, Array a2);
int compare(const void* a, const void* b);

//CPU BASELINE/NAIVE
void LCM_cpu_baseline(igraph_t &graph);

//CUDA ERROR
void checkCudaError(hipError_t e, const char* in) {
	if (e != hipSuccess) {
		printf("CUDA Error: %s, %s \n", in, hipGetErrorString(e));
		//exit(EXIT_FAILURE);
	}
}

//MAIN
int main(int argc, char** argv) {
	//checks arguments
	if(argc < 3) {

		printf("\nToo few arguments. Usage: $ %s graphFile all/out\n", argv[0]);
		return -1;
	}

	//graph direction out or all
	if(!strcmp(argv[2], "out"))
		OUTALL = IGRAPH_OUT;
	else if(!strcmp(argv[2], "all"))
		OUTALL = IGRAPH_ALL;
	else {
		printf("\nInvalid Graph Direction. Use out or all.\nUsage: ./%s graphFile all/out\n", argv[0]);
	}
	
	//cpu timing shit
	struct timeval stop, start;
	
	//opens graph file passed as 1st argument
	FILE *inputFile;
	inputFile = fopen(argv[1], "r");
	if(inputFile == NULL)
	{
		printf("Could not load input file...\n");
		return 1;
	}
	
	//graph var and builds graph from file
	igraph_t graph;
	igraph_read_graph_ncol(&graph, inputFile, NULL, true, IGRAPH_ADD_WEIGHTS_NO, IGRAPH_DIRECTED);
	int n_vertices = igraph_vcount(&graph);

	//cpu naive, needs tons of host memory and really slow
	// if(n_vertices < 20000) {
	// 	gettimeofday(&start, NULL);
	// 	LCM_cpu_baseline(graph);
	// 	gettimeofday(&stop, NULL);
	// 	printf("CPU Naive Running Time on %d Nodes: %2f sec\n", n_vertices, ((stop.tv_sec - start.tv_sec) * 1000.0f + (stop.tv_usec - start.tv_usec) / 1000.0f) / 1000.0f);
	// }
	// else
	// 	printf("\nCPU Naive cant run %d nodes.\n", n_vertices);


	//CPU OPTIMIZED, COMMENT OUT TO SKIP
	if(!argv[3]) {
		printf("Running CPU Optimized Single Thread Code\n");
		gettimeofday(&start, NULL);
		LCM_CPU(graph, OUTALL);
		gettimeofday(&stop, NULL);
		printf("CPU Optimized Running Time on %d Nodes: %2f sec\n", n_vertices, ((stop.tv_sec - start.tv_sec) * 1000.0f + (stop.tv_usec - start.tv_usec) / 1000.0f) / 1000.0f);
	}
	else
		printf("CPU Optimized not running.\n");

	int threads = 128;
	// //GPU KERNELS, UNCOMMENT TO USE, COMMENT OUT TO SKIP
	// if(n_vertices < 20000) {
	// 	Naive_Prep(graph);
	// 	OPT_1_PREP(graph);
	// }
	// else
	// 	printf("\nGPU NAIVE & OPT_1 cant run %d nodes.\n", n_vertices);

	//OPT_2_PREP(graph, OUTALL, threads);
	//OPT_3_PREP(graph);
	printf("Running GPU Optimized");
	OPT_4_PREP(graph, OUTALL, threads);
	
	return 0;
}

//NAIVE GPU
void Naive_Prep(igraph_t &graph) {

	//creates adjacency matrix and gets num vertices
	int *matrix, n_vertices = igraph_vcount(&graph);
	long int vsize;
	
	//vertice adj vectors, intialized to size 0
	igraph_vector_t vec;
	igraph_vector_init(&vec, 0);
	
	//initializes matrix and sets to zero
	matrix = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(matrix, 0, sizeof(int)*n_vertices*n_vertices);

	//builds adj matrix
	for(int i = 0; i < n_vertices; i++) {
		
		//gets vertice i's neighbors and number of adjacencies
		igraph_neighbors(&graph, &vec, i, OUTALL);
		vsize = igraph_vector_size(&vec);

		//puts ones in the adj matrix where they belong
		for(int j = 0; j < vsize; j++) {

			matrix[i*n_vertices + (int)VECTOR(vec)[j]] = 1;
		}
	}

	//CUDA SHIT
	int hsize = 64;
	int *hist, *d_hist;
	hist = (int*)malloc(sizeof(int)*hsize);
	hipMalloc((void**)&d_hist, sizeof(int)*hsize);

	//creates 2 adjacency matrix graphs for device
	int *d_matrix, *d_result;
	hipMalloc((void**)&d_matrix, sizeof(int)*n_vertices*n_vertices);
	hipMalloc((void**)&d_result, sizeof(int)*n_vertices*n_vertices);
	
	//copys adj matrix to device and sets device hist and result to zero
	hipMemcpy(d_matrix, matrix, sizeof(int)*n_vertices*n_vertices, hipMemcpyHostToDevice);
	hipMemset(d_result, 0, sizeof(int)*n_vertices*n_vertices);
	hipMemset(d_hist, 0, sizeof(int)*hsize);
	//memset(hist, 0, sizeof(int)*hsize);

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//kernels for lcm and hist
	Naive<<<n_vertices, 1024>>>(d_matrix, d_result, n_vertices);
	Naive_Hist<<<n_vertices, 1024>>>(d_result, d_hist, n_vertices);
	
	//copies hist back to host
	checkCudaError(hipMemcpy(hist, d_hist, sizeof(int)*hsize, hipMemcpyDeviceToHost), "D_HIST TO HOST");
	
	//kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//prints gpu histogram
	printf("\nGPU Naive HISTOGRAM\n");
	for(int i = 1; i < hsize; i++) {
		if ((hist[i] / i) > 0)
			printf("%d    %d\n", i, (hist[i] / i));
	}

	//prints kernel running time
	//printf("\n******** Total Running Time of Kernel = %0.5f ms *******\n", elapsedTime);
	printf("\n******** Naive Processed %d Node Graph In %0.5f sec *******\n", n_vertices, elapsedTime/1000);

	//frees all the shit
	free(matrix);
	free(hist);
	hipFree(d_matrix);
	hipFree(d_result);
	hipFree(d_hist);
}

//uses adjaceny matrix, slow and takes a shit load of device memory, lots of zeros
__global__ void Naive(int* d_matrix, int* d_result, int n_vertices) {

	//each block takes care of a whole row
	//columns to be compared to same row are threads
	int row = blockIdx.x;
	int col = threadIdx.x;
	int cval;

	//compares vertice blockIdx.x to all other vertices, increments by blockDim
	if(row < n_vertices && col < n_vertices)
	for(int i = col; i < n_vertices; i += blockDim.x) {

		//sets graphs horizontal to 0
		if(row == i) {
			d_result[row*n_vertices + i] = 0;
			continue;
		}

		//sets to zero
		cval = 0;

		//gets row x col
		for(int j = 0; j < n_vertices; j++)
			cval += d_matrix[row*n_vertices + j] * d_matrix[n_vertices*j + i];

		//puts cval into graph
		d_result[row*n_vertices + i] = cval;
	}
	
	//syncs threads so new row is done and sorts it using thrust on thread 0
	__syncthreads();
	if(col == 0 && row < n_vertices)
		thrust::sort(thrust::device, &d_result[row*n_vertices], &d_result[row*n_vertices] + n_vertices);
}

//builds histogram, lots of zeros
__global__ void Naive_Hist(int* d_result, int* d_hist, int n_vertices) {

	//each block compares the same row to all others row2
	int row = blockIdx.x;
	int row2 = threadIdx.x;
	bool equal;

	//shared count for whole block/same vertice
	__shared__ int count;

	//one thread sets count to zero and syncsthreads.
	if(row2 == 0)
		count = 0;
	__syncthreads();

	//checks equality to other vertices
	if(row < n_vertices && row2 < n_vertices)
	for(int i = row2; i < n_vertices; i += blockDim.x) {

		//checks equality of vertices lcm
		equal = false;
		for(int j = 0; j < n_vertices; j++) {

			if(d_result[row*n_vertices +j] == d_result[i*n_vertices + j])
				equal = true;
			else {
				equal = false;
				break;
			}
		}

		//adds to count if vertices are equal
		if(equal)
			atomicAdd(&count, 1);
	}

	//syncsthreads so count is done and increments hist[count]
	__syncthreads();
	if(row < n_vertices && row2 == 0 && count > 0)
		atomicAdd(&d_hist[count], 1);
}

//OPT 1 PREP & KERNEL
void OPT_1_PREP(igraph_t &graph) {

	//num vertices
	int n_vertices = igraph_vcount(&graph);

	//1D adj list graphs and sizes
	int *adj;
	int *adjsizes = (int*)malloc(sizeof(int)*(n_vertices + 1));

	//vector for single vertices adj list
	igraph_vector_t neisVec;
	igraph_vector_init(&neisVec, 0);

	//figures out threads per block
	int threads;
	if(n_vertices > 1024)
		threads = 1024;
	else
		threads = n_vertices;

	//gets each vertex's number of neighbors and total neighbors
	adjsizes[0] = 0;
	for(int i = 1; i <= n_vertices; i++) {

		igraph_neighbors(&graph, &neisVec, i-1, OUTALL);
		adjsizes[i] = igraph_vector_size(&neisVec) + adjsizes[i-1];

	}

	

	//creats jagged & flattened to 1D adj list	
	adj = (int*)malloc(sizeof(int)*adjsizes[n_vertices]);

	//creates 1d adj list
	for(int i = 0; i < n_vertices; i++) {

		//gets neighbors and number of neighbors
		igraph_neighbors(&graph, &neisVec, i, OUTALL);

		//loads in vertice i's adjancent neighbors
		//printf("\n%d: ", i);
		for(int j = 0; j < adjsizes[i+1] - adjsizes[i]; j++) {
			
			adj[adjsizes[i] + j] = (int)VECTOR(neisVec)[j];

			//printf("[%d, %d] ", adj[adjsizes[i] + j], (int)VECTOR(neisVec)[j]);
		}
	}



	//device vars
	int *d_adj, *d_lcm, *d_adjsizes, *d_hist;

	//histogram vars
	int *hist;
	hist = (int*)malloc(sizeof(int)*n_vertices);
	memset(hist, 0, sizeof(int)*n_vertices);

	//mallocs and copys
	checkCudaError(hipMalloc((void**)&d_adj, sizeof(int)*adjsizes[n_vertices]), "Malloc d_adj");
	checkCudaError(hipMalloc((void**)&d_adjsizes, sizeof(int)*(n_vertices+1)), "Malloc d_adjsizes");
	checkCudaError(hipMalloc((void**)&d_lcm, sizeof(int)*n_vertices*n_vertices), "Malloc d_lcm");

	//copys adj list to device and initializes lcm to zero
	checkCudaError(hipMemcpy(d_adj, adj, sizeof(int)*adjsizes[n_vertices], hipMemcpyHostToDevice), "Memcpy d_adj");
	checkCudaError(hipMemcpy(d_adjsizes, adjsizes, sizeof(int)*(n_vertices+1), hipMemcpyHostToDevice), "Memcpy d_adjsizes");
	checkCudaError(hipMemset(d_lcm, 0, sizeof(int)*n_vertices*n_vertices), "Memset d_lcm");

	//device histogram stuff
	checkCudaError(hipMalloc((void**)&d_hist, sizeof(int)*n_vertices), "Malloc d_hist");
	checkCudaError(hipMemset(d_hist, 0, sizeof(int)*n_vertices), "Memset d_hist");

	//SIZE OF SHIT
	//printf("\nSize(adj) =     %ld Bytes\nSize(adjsize) = %ld Bytes\nSize(hist) =    %ld Bytes\nSize(lcm) =     %ld Bytes", sizeof(int)*adjsizes[n_vertices], sizeof(int)*(n_vertices + 1), sizeof(int)*n_vertices, sizeof(int)*n_vertices*n_vertices);

	

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//kernel call
	OPT_1<<<n_vertices, threads>>>(d_adj, d_lcm, d_adjsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, Test Kernel Launch");
	// printf("\nTEST\n");
	//hipDeviceSynchronize();
	

	//DEBUG
	// int *lcm = (int*)malloc(sizeof(int)*n_vertices*n_vertices);
	// hipMemcpy(lcm, d_lcm, sizeof(int)*n_vertices*n_vertices, hipMemcpyDeviceToHost);
	// for(int i = 0; i < n_vertices; i++) {

	// 	printf("\nv%d: ", i);
	// 	for(int j = 0; j < n_vertices; j++) {

	// 		printf("%d ", lcm[i*n_vertices + j]);
	// 	}
	// 	printf("\n");
	// }
	// for(int i = 0; i < n_vertices; i++) {

	// 	int count = 0;

	// 	for(int j = 0; j < n_vertices; j++) {

	// 		bool equal = false;

	// 		for(int k = 0; k < n_vertices; k++) {

	// 			if(lcm[i*n_vertices + k] == lcm[j*n_vertices + k])
	// 				equal = true;
	// 			else {
	// 				equal = false;
	// 				break;
	// 			}
	// 		}

	// 		if(equal)
	// 			++count;
	// 	}
	// 	// if(countMax < count)
	// 	// 		countMax = count;

	// 	++hist[count];
	// }


	// histogram shit
	
	OPT_1_HIST<<<n_vertices, threads>>>(d_lcm, d_hist, n_vertices);

	//kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	checkCudaError(hipGetLastError(), "Checking Last Error, Test Hist Launch");
	checkCudaError(hipMemcpy(hist, d_hist, sizeof(int)*n_vertices, hipMemcpyDeviceToHost), "Memcpy d_hist to host");

	//prints gpu histogram
	printf("\nGPU OPT_1 HISTOGRAM\n");
	for(int i = 1; i < n_vertices; i++) {
		if ((hist[i] / i) > 0)
			printf("%d    %d\n", i, (hist[i] / i));
	}

	//prints kernel running time
	//printf("\n******** Total Running Time of Kernel = %0.5f ms *******\n", elapsedTime);
	printf("******** OPT_1 Processed %d Node Graph In %0.5f sec *******\n", n_vertices, elapsedTime/1000);

	//frees everything
	hipFree(d_hist);
	hipFree(d_lcm);
	hipFree(d_adj);
	hipFree(d_adjsizes);
	free(hist);
	free(adj);
	free(adjsizes);
}	

//OPTIMIZATION 1
__global__ void OPT_1(int* adj, int* lcm, int* sizes, int n) {
	
	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	int cval;

	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			lcm[vertex*n + i] = 0;
			continue;
		}

		//resets count
		cval = 0;

		//for loop that goes through vertex neighbors
		for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

			//loop compares to other vertex i/vcomp
			for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

				if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

					++cval;
					break;
				}
			}
		}

		//puts in lcm
		lcm[vertex*n + i] = cval;
	}

	//sorts vertex lcm once block is done
	__syncthreads();
	if(vcomp == 0 && vertex < n)
		thrust::sort(thrust::device, &lcm[vertex*n], &lcm[vertex*n] + n);
}

__global__ void OPT_1_HIST(int* lcm, int* hist, int n) {

	//
	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	bool equal;
	
	//
	__shared__ int cval;

	//
	if(vcomp == 0)
		cval = 0;
	__syncthreads();

	//
	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			atomicAdd(&cval, 1);
			continue;
		}
		
		equal = false;

		for(int j = 0; j < n; j++) {

			if(lcm[vertex*n + j] == lcm[i*n + j])
				equal = true;
			
			else {
				equal = false;
				break;
			}
		}

		if(equal)
			atomicAdd(&cval, 1);
	}

	__syncthreads();
	if(vertex < n && vcomp == 0 && cval > 0) {
		atomicAdd(&hist[cval], 1);
		//printf("\nv%d: %d\n", vertex, cval);
	}
}

//OPTIMIZED 2 SATHYAS GPU 1
int OPT_2_PREP(igraph_t &graph, igraph_neimode_t OUTALL, int numThreads)
{
    //printf("\nAllocating Adjacency List\n");
    int n_vertices = igraph_vcount(&graph);
    igraph_adjlist_t al;
    igraph_adjlist_init(&graph, &al, OUTALL);
    igraph_adjlist_simplify(&al);

    int **adjList2D;
    int totalSize = 0;

    int *adjList, *d_adjList;
    int *sizeAdj, *d_sizeAdj;

    int *lcmMatrix, *d_lcmMatrix;

    int *d_LCMSize, *LCMSize, *LCMSize_Calc;
    
    adjList2D = (int **) calloc(n_vertices, sizeof(int *));
    sizeAdj = (int *) malloc(n_vertices * sizeof(int));
    LCMSize = (int *) malloc(n_vertices * sizeof(int));
    LCMSize_Calc = (int *) malloc(n_vertices * sizeof(int));
    memset(LCMSize, 0, n_vertices*sizeof(int));
    memset(LCMSize_Calc, 0, n_vertices*sizeof(int));
    //printf("Computing Adjacency List - %d vertices...\n", n_vertices);

    for (int i = 0; i < n_vertices; i++) {
        igraph_vector_int_t *adjVec = igraph_adjlist_get(&al, i);

        // igraph_vector_t adjVec;
        // igraph_vector_init(&adjVec, 0);
        // igraph_neighbors(&graph, &adjVec, i, OUTALL);

        adjList2D[i] = (int *) malloc(igraph_vector_int_size(adjVec) * sizeof(int));
        sizeAdj[i] = (int) igraph_vector_int_size(adjVec);
        totalSize += sizeAdj[i];
        for(int k = 0; k< igraph_vector_int_size(adjVec); k++)
        {
            adjList2D[i][k] = (int) VECTOR(*adjVec)[k];
        }
    }

    for(int i = 0; i< n_vertices; i++)
    {
        qsort(adjList2D[i], sizeAdj[i], sizeof(int), compare);
    }
    
    adjList = (int *) malloc(totalSize * sizeof(int));
    int l = -1;
    for (int q = 0; q < n_vertices; q++)
    {
        for (int t = 0; t < sizeAdj[q]; t++)
        {
            l++;
            adjList[l] = adjList2D[q][t];
        }
    }
    for(int i = 0; i< n_vertices; i++)
    {
        free(adjList2D[i]);
        if(i>0)
        {
            sizeAdj[i] += sizeAdj[i - 1];
        }
    }
    
    //kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    free(adjList2D);
    // memset(LCMSize, 0, n_vertices*sizeof(int));
    //printf("%d-%d\n", totalSize, sizeAdj[n_vertices-1]);
    //printf("Got Adj List...\n Allocating on gpu mem...");
    checkCudaError(hipMalloc((void**)&d_adjList, totalSize * sizeof(int)), "Malloc Error d_adjList");
    checkCudaError(hipMalloc((void**)&d_sizeAdj, n_vertices * sizeof(int)), "Malloc Error d_sizeAdj");
    checkCudaError(hipMalloc((void**)&d_LCMSize, n_vertices * sizeof(int)), "Malloc Error d_sizeAdj");

    hipMemcpy(d_adjList, adjList, totalSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sizeAdj, sizeAdj, n_vertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_LCMSize, LCMSize_Calc, n_vertices * sizeof(int), hipMemcpyHostToDevice);

    dim3 DimGrid(ceil(n_vertices/numThreads), 1, 1);   
    if (n_vertices%numThreads) 
    {
        DimGrid.x++;
    }

    dim3 DimBlock(numThreads, 1, 1);
    int totLCMSize = 0;
    //printf("Launching Size Kernel...\n");
    OPT_2_SIZES<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_LCMSize, n_vertices);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Checking Last Error, Size Kernel Launch");
    hipMemcpy(LCMSize_Calc, d_LCMSize, n_vertices * sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i = 0; i<n_vertices; i++)
    {
        totLCMSize += LCMSize_Calc[i];
        LCMSize[i] = LCMSize_Calc[i];
    }

    for(int i = 1; i<n_vertices; i++)
    {
        if(i>0)
            LCMSize[i] += LCMSize[i - 1];
    }
    //printf("%d - %d\n", totalSize, totLCMSize);
    
    lcmMatrix = (int *) malloc(totLCMSize * sizeof(int));
    memset(lcmMatrix, 0, totLCMSize*sizeof(int));
    checkCudaError(hipMalloc((void**)&d_lcmMatrix, totLCMSize * sizeof(int)), "Malloc Error d_lcmMatrix");
    hipMemcpy(d_lcmMatrix, lcmMatrix, totLCMSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_LCMSize, LCMSize, n_vertices * sizeof(int), hipMemcpyHostToDevice);
    //printf("Launching LCM Kernel...\n");
    
    // LCM_Kernel<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_lcmMatrix, d_LCMSize, n_vertices);
    OPT_2<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_lcmMatrix, d_LCMSize, n_vertices);
    
	hipDeviceSynchronize();
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Checking Last Error, Kernel Launch");
    //printf("Copying to CPU Memory...\n");
    checkCudaError(hipMemcpy(lcmMatrix, d_lcmMatrix, totLCMSize * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy Error d_lcmMatrix");
    // checkCudaError(hipMemcpy(LCMSize, d_LCMSize, n_vertices * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy Error LCMSize");
    
    hipFree(d_lcmMatrix);
    hipFree(d_LCMSize);
    hipFree(d_adjList);
    hipFree(d_sizeAdj);
    free(sizeAdj);
    free(adjList);

    //printf("Allocating Histogram...\n");
    int *histo;
    histo = (int *) malloc(n_vertices * sizeof(int));
    memset(histo, 0, sizeof(int)*n_vertices);
    int count = 0, countMax = -1;
    // int *neisVec1, *neisVec2;
    // neisVec1 = (int *) malloc(1 * sizeof(int));
    // neisVec2 = (int *) malloc(1 * sizeof(int));
   
    //printf("Sorting LCM...\n");
    // int totLCMSize1 = 0;
    
	for(int i = 0; i< n_vertices; i++)
	{
		int offset = 0;
		if(i > 0)
		{
			offset = LCMSize[i - 1];
		}
		// printf("%d - %d\n", offset, LCMSize_Calc[i]);
		qsort(lcmMatrix + offset, LCMSize_Calc[i], sizeof(int), compare);
		// totLCMSize1 += LCMSize[i];
	}
	// for(int i = 0; i<LCMSize_Calc[4000]; i++)
	// 	printf("%d-", lcmMatrix[LCMSize[3999] + i]);
	//printf("Computing Histogram...\n");
	// return 0;
    for(int i = 0; i< n_vertices; i++)
    {
        int iStart = 0;
        if(i>0)
            iStart = LCMSize[i - 1]; //Offset
        count = 0;

        for(int j = 0; j < n_vertices; j++) {
            if(LCMSize_Calc[i] != LCMSize_Calc[j])
                continue;
            
            int jStart = 0;
            
            if(j>0)
                jStart = LCMSize[j - 1]; //Offset
            
            int eq = 1;
            for(int k = 0; k < LCMSize_Calc[i]; k++)
            {
            	if(lcmMatrix[iStart + k] != lcmMatrix[jStart + k])
            	{
            		eq = 0;
            		break;
            	}
            }
            if(eq == 1)
            {               
                count++;
            }
        }

        if(countMax < count)
            countMax = count;
        histo[count]++;
    }

    //kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);


    printf("\nGPU OPT_2 HISTOGRAM\n");
    for(int i = 1; i <= countMax; i++) {
        if ((long) (histo[i] / i) > 0)
            printf("%d    %d\n", i, (int) (histo[i] / i));
    }

    //kernels total times
	//printf("\n******** Processed %d Node Graph In %0.5f ms *******\n", n_vertices, elapsedTime);
	printf("******** OPT_2 Processed %d Node Graph In %0.5f sec *******\n", n_vertices, elapsedTime/1000);

    //frees
    free(lcmMatrix);
    free(LCMSize_Calc);
    free(LCMSize);
    free(histo);
	return 0;
}

__global__ void OPT_2_SIZES(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;     
    if(i<n_vertices)
    {
        int indexUsed = 0;
        int iStart = 0, iEnd = 0;
        int k = 0;

        if(i > 0)
        {       
            k = d_sizeAdj[i-1];
        }

        iEnd = d_sizeAdj[i];

        __syncthreads();

        for(int j = 0; j < n_vertices; j++) {
            if(i==j)
                continue;
            iStart = k;
            int jStart = 0, jEnd = 0;

            if(j > 0)
                jStart = d_sizeAdj[j-1];
            jEnd = d_sizeAdj[j];
            
            int compVec = 0;

            while (iStart < iEnd && jStart < jEnd)
            {
                    if(d_adjList[iStart] < d_adjList[jStart])
                        iStart++;
                    else if (d_adjList[jStart] < d_adjList[iStart])
                        jStart++;
                    else // if arr1[i] == arr2[j] 
                    {
                        jStart++;
                        iStart++;
                        compVec++;
                        // break;
                    }
            }

            if (compVec > 0)
            {
                indexUsed++;
            }
        }
    
        __syncthreads();
        d_LCMSize[i] = indexUsed;
        // __syncthreads();
    
    }

}

__global__ void OPT_2(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;     
    if(i<n_vertices)
    {
        int indexUsed = 0, indexOffset = 0;
        int iStart = 0, iEnd = 0;
        int k = 0;

        if(i > 0)
        {       
            k = d_sizeAdj[i-1];
            indexOffset = d_LCMSize[i-1];
        }

        iEnd = d_sizeAdj[i];
        
        for(int j = indexOffset; j<iEnd; j++)
        {
            d_lcmMatrix[j] = 0;
        }

        __syncthreads();

        for(int j = 0; j < n_vertices; j++) {
            if(i==j)
                continue;
            iStart = k;
            int jStart = 0, jEnd = 0;

            if(j > 0)
                jStart = d_sizeAdj[j-1];
            jEnd = d_sizeAdj[j];
            
            int compVec = 0;

            while (iStart < iEnd && jStart < jEnd)
            {
                    if(d_adjList[iStart] < d_adjList[jStart])
                        iStart++;
                    else if (d_adjList[jStart] < d_adjList[iStart])
                        jStart++;
                    else // if arr1[i] == arr2[j] 
                    {
                        jStart++;
                        iStart++;
                        compVec++;
                    }
            }

            if (compVec > 0)
            {
                atomicAdd((int*)&d_lcmMatrix[indexUsed + indexOffset], compVec);
                // d_lcmMatrix[indexUsed + indexOffset] = compVec;
                indexUsed++;
            }
            // __syncthreads();
        }
    
        // __syncthreads();
        // d_LCMSize[i] = indexUsed;
        // __syncthreads();
    
    }

}

//OPTIMIZATION 2 KERNELS & PREP
void OPT_3_PREP(igraph_t &graph) {

	//num vertices
	int n_vertices = igraph_vcount(&graph);

	//1D adj list graphs and sizes
	int *adj;
	int *adjsizes = (int*)malloc(sizeof(int)*(n_vertices + 1));
	int lcmsizes;

	//vector for single vertices adj list
	igraph_vector_t neisVec;
	igraph_vector_init(&neisVec, 0);

	//adj list shit
	igraph_adjlist_t al;
    igraph_adjlist_init(&graph, &al, OUTALL);
    igraph_adjlist_simplify(&al);
    igraph_vector_int_t *adjVec;

	//figures out threads per block
	int threads_max = 128;
	int threads;
	if(n_vertices > threads_max)
		threads = threads_max;
	else
		threads = n_vertices;

	//histogram vars
	int *hist;
	hist = (int*)malloc(sizeof(int)*n_vertices);
	memset(hist, 0, sizeof(int)*n_vertices);

	//gets each vertex's number of neighbors and total neighbors
	adjsizes[0] = 0;
	for(int i = 1; i <= n_vertices; i++) {

		// igraph_neighbors(&graph, &neisVec, i-1, OUTALL);
		// adjsizes[i] = igraph_vector_size(&neisVec) + adjsizes[i-1];

		adjVec = igraph_adjlist_get(&al, i-1);
		adjsizes[i] = igraph_vector_int_size(adjVec) + adjsizes[i-1];

	}

	//creats jagged & flattened to 1D adj list	
	adj = (int*)malloc(sizeof(int)*adjsizes[n_vertices]);

	//creates 1d adj list
	for(int i = 0; i < n_vertices; i++) {

		//gets neighbors and number of neighbors
		//igraph_neighbors(&graph, &neisVec, i, OUTALL);

		//loads in vertice i's adjancent neighbors
		// for(int j = 0; j < adjsizes[i+1] - adjsizes[i]; j++)
		// 	adj[adjsizes[i] + j] = (int)VECTOR(neisVec)[j];

		adjVec = igraph_adjlist_get(&al, i);

		for(int j = 0; j < adjsizes[i+1] - adjsizes[i]; j++)
			adj[adjsizes[i] + j] = (int)VECTOR(*adjVec)[j];
	}

	//device vars
	int *d_adj, *d_lcm, *d_adjsizes, *d_lcmsizes, *d_hist, *d_lcm_max;

	//kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//mallocs device shit
	checkCudaError(hipMalloc((void**)&d_adj, sizeof(int)*adjsizes[n_vertices]), "Malloc d_adj");
	checkCudaError(hipMalloc((void**)&d_adjsizes, sizeof(int)*(n_vertices+1)), "Malloc d_adjsizes");
	checkCudaError(hipMalloc((void**)&d_lcmsizes, sizeof(int)*(n_vertices+1)), "Malloc d_lcmsizes");
	checkCudaError(hipMalloc((void**)&d_lcm_max, sizeof(int)), "Malloc d_lcm_max");

	//copys adj list to device and initializes lcm to zero
	checkCudaError(hipMemcpy(d_adj, adj, sizeof(int)*adjsizes[n_vertices], hipMemcpyHostToDevice), "Memcpy d_adj");
	checkCudaError(hipMemcpy(d_adjsizes, adjsizes, sizeof(int)*(n_vertices+1), hipMemcpyHostToDevice), "Memcpy d_adjsizes");
	checkCudaError(hipMemset(d_lcmsizes, 0, sizeof(int)*(n_vertices+1)), "Memset d_lcmsizes");
	//checkCudaError(hipMemset(&d_lcm_max, 0, sizeof(int)), "Memset d_lcmsizes");

	

	//SIZE OF SHIT
	//printf("\nSize(adj) =     %ld Bytes\nSize(adjsize) = %ld Bytes\nSize(hist) =    %ld Bytes\nSize(lcm) =     %ld Bytes", sizeof(int)*adjsizes[n_vertices], sizeof(int)*(n_vertices + 1), sizeof(int)*n_vertices, sizeof(int)*n_vertices*n_vertices);

	//lcm sizes kernel
	OPT_3_SIZES<<<n_vertices, threads>>>(d_adj, d_lcmsizes, d_adjsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_SIZES Kernel Launch");
	//hipDeviceSynchronize();
	OPT_3_SIZES_SUM<<<1,1>>>(d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_SIZES_SUM Kernel Launch");
	//hipDeviceSynchronize();

	// //kernel execution stop
	// hipEventRecord(stop, 0);
	// hipEventSynchronize(start);
	// hipEventSynchronize(stop);
	// hipEventElapsedTime(&elapsedTime, start, stop);
	// hipEventDestroy(start);
	// hipEventDestroy(stop);

	//creates lcm adj list shit
	checkCudaError(hipMemcpy(&lcmsizes, &d_lcmsizes[n_vertices], sizeof(int), hipMemcpyDeviceToHost), "Memcpy d_lcmsizes to lcmsizes");
	checkCudaError(hipMalloc((void**)&d_lcm, sizeof(int)*lcmsizes), "Malloc d_lcm");
	checkCudaError(hipMemset(d_lcm, 0, sizeof(int)*lcmsizes), "Memset d_lcm");

	// //kernel execution time crap 2
	// float elapsedTime3;
	// hipEvent_t start3, stop3;
	// hipEventCreate(&start3);
	// hipEventCreate(&stop3);
	// hipEventRecord(start3, 0);

	//get lcm shit
	OPT_3<<<n_vertices, threads>>>(d_adj, d_lcm, d_adjsizes, d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3 Kernel Launch");

	//DEBUG HIST
	// int *lcm = (int*)malloc(sizeof(int)*lcmsizes);
	// int *lsizes = (int*)malloc(sizeof(int)*(n_vertices+1));
	// hipMemcpy(lcm, d_lcm, sizeof(int)*lcmsizes, hipMemcpyDeviceToHost);
	// hipMemcpy(lsizes, d_lcmsizes, sizeof(int)*(n_vertices+1), hipMemcpyDeviceToHost);
	// for(int i = 0; i < n_vertices; i++) {

	// 	int count = 0;

	// 	for(int j = 0; j < n_vertices; j++) {

	// 		if(lsizes[i+1] - lsizes[i] != lsizes[j+1] - lsizes[j])
	// 			continue;
			
	// 		bool equal = false;

	// 		for(int k = 0; k < lsizes[i+1] - lsizes[i]; k++) {

	// 			if(lcm[lsizes[i] + k] == lcm[lsizes[j] + k])
	// 				equal = true;
	// 			else {
	// 				equal = false;
	// 				break;
	// 			}
	// 		}

	// 		if(equal)
	// 			++count;
	// 	}

	// 	++hist[count];
	// }
	// free(lcm);
	// free(lsizes);

	//histogram
	hipFree(d_adj);
	hipFree(d_adjsizes);
	checkCudaError(hipMalloc((void**)&d_hist, sizeof(int)*n_vertices), "Malloc d_hist");
	checkCudaError(hipMemset(d_hist, 0, sizeof(int)*n_vertices), "Memset d_hist");
	OPT_3_HIST<<<n_vertices, threads>>>(d_lcm, d_hist, d_lcmsizes, n_vertices);
	checkCudaError(hipGetLastError(), "Checking Last Error, OPT_3_HIST Kernel Launch");
	checkCudaError(hipMemcpy(hist, d_hist, sizeof(int)*n_vertices, hipMemcpyDeviceToHost), "D_HIST TO HOST");

	//kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//prints gpu histogram
	printf("\nGPU OPT_3 HISTOGRAM\n");
	for(int i = 1; i < n_vertices; i++) {
		if ((hist[i] / i) > 0)
			printf("%d    %d\n", i, (hist[i] / i));
	}

	//kernels total times
	//printf("\n******** Processed %d Node Graph In %0.5f ms *******\n", n_vertices, elapsedTime);
	printf("******** OPT_3 Processed %d Node Graph In %0.5f sec *******\n", n_vertices, elapsedTime/1000);

	//frees all the shit
	free(adj);
	free(hist);
	free(adjsizes);
	hipFree(d_hist);
	hipFree(d_lcm);
	hipFree(d_lcmsizes);
}

__global__ void OPT_3_SIZES(int* adj, int* lcmsizes, int* sizes, int n) {

	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	int cval;

	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		//skips to next vertex
		if(vertex == i) {
			continue;
		}

		//resets count
		cval = 0;

		//for loop that goes through vertex neighbors
		for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

			//loop compares to other vertex i/vcomp
			for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

				if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

					++cval;
					break;
				}
			}

			if(cval > 0) {
				atomicAdd(&lcmsizes[vertex + 1], 1);
				break;
			}
		}
	}
}

//
__global__ void OPT_3_SIZES_SUM(int* lcmsizes, int n) {
	
	for(int i = 0; i < n; i++)
		lcmsizes[i+1] += lcmsizes[i];
}

//
__global__ void OPT_3(int* adj, int* lcm, int* sizes, int* lcmsizes, int n) {

	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	int cval;

	__shared__ int pos;

	if(vcomp == 0)
		pos = 0;
	__syncthreads();

	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			continue;
		}

		//resets count
		cval = 0;

		//for loop that goes through vertex neighbors
		for(int j = 0; j < sizes[vertex + 1] - sizes[vertex]; j++) {

			//loop compares to other vertex i/vcomp
			for(int k = 0; k < sizes[i+1] - sizes[i]; k++) {

				if(adj[sizes[vertex] + j] == adj[sizes[i] + k]) {

					++cval;
					break;
				}
			}
		}

		//copies to array
		if(cval > 0) {
			//__syncthreads();
			lcm[lcmsizes[vertex] + atomicAdd(&pos, 1)] = cval;
		}
	}

	//sorts vertex lcm once block is done
	__syncthreads();
	if(vcomp == 0 && vertex < n)
		thrust::sort(thrust::device, &lcm[lcmsizes[vertex]], &lcm[lcmsizes[vertex+1]]);
}

//
__global__ void OPT_3_HIST(int* lcm, int* hist, int* lcmsizes, int n) {

	//
	int vertex = blockIdx.x;
	int vcomp = threadIdx.x;
	bool equal;
	
	//
	__shared__ int cval;

	//
	if(vcomp == 0)
		cval = 0;
	__syncthreads();

	//
	if(vertex < n && vcomp < n)
	for(int i = vcomp; i < n; i += blockDim.x) {

		if(vertex == i) {
			atomicAdd(&cval, 1);
			continue;
		}

		if((lcmsizes[vertex+1] - lcmsizes[vertex]) != (lcmsizes[i+1] - lcmsizes[i]))
			continue;
		
		equal = false;

		for(int j = 0; j < lcmsizes[vertex+1] - lcmsizes[vertex]; j++) {

			if(lcm[lcmsizes[vertex] + j] == lcm[lcmsizes[i] + j])
				equal = true;
			
			else {
				equal = false;
				break;
			}
		}

		if(equal)
			atomicAdd(&cval, 1);
	}

	__syncthreads();
	if(vertex < n && vcomp == 0 && cval > 0) {
		atomicAdd(&hist[cval], 1);
		//printf("\nv%d: %d\n", vertex, cval);
	}
}

//OPTIMIZED 4 SATHYAS GPU 2
int OPT_4_PREP(igraph_t &graph, igraph_neimode_t OUTALL, int numThreads)
{
    //printf("\nAllocating Adjacency List\n");
    int n_vertices = igraph_vcount(&graph);
    igraph_adjlist_t al;
    igraph_adjlist_init(&graph, &al, OUTALL);
    igraph_adjlist_simplify(&al);

    int **adjList2D;
    int totalSize = 0;

    int *adjList, *d_adjList;
    int *sizeAdj, *d_sizeAdj;

    int *lcmMatrix, *d_lcmMatrix;

    int *d_LCMSize, *LCMSize, *LCMSize_Calc;

    igraph_vector_int_t *adjVec;
    
    adjList2D = (int **) calloc(n_vertices, sizeof(int *));
    sizeAdj = (int *) malloc(n_vertices * sizeof(int));
    LCMSize = (int *) malloc(n_vertices * sizeof(int));
    LCMSize_Calc = (int *) malloc(n_vertices * sizeof(int));
    memset(LCMSize, 0, n_vertices*sizeof(int));
    memset(LCMSize_Calc, 0, n_vertices*sizeof(int));
    //printf("Computing Adjacency List - %d vertices...\n", n_vertices);

    for (int i = 0; i < n_vertices; i++) {
        adjVec = igraph_adjlist_get(&al, i);

        // igraph_vector_t adjVec;
        // igraph_vector_init(&adjVec, 0);
        // igraph_neighbors(&graph, &adjVec, i, OUTALL);

        adjList2D[i] = (int *) malloc(igraph_vector_int_size(adjVec) * sizeof(int));
        sizeAdj[i] = (int) igraph_vector_int_size(adjVec);
        totalSize += sizeAdj[i];
        for(int k = 0; k< igraph_vector_int_size(adjVec); k++)
        {
            adjList2D[i][k] = (int) VECTOR(*adjVec)[k];
        }
    }

    for(int i = 0; i< n_vertices; i++)
    {
        qsort(adjList2D[i], sizeAdj[i], sizeof(int), compare);
    }
    
    adjList = (int *) malloc(totalSize * sizeof(int));
    int l = -1;
    for (int q = 0; q < n_vertices; q++)
    {
        for (int t = 0; t < sizeAdj[q]; t++)
        {
            l++;
            adjList[l] = adjList2D[q][t];
        }
    }
    for(int i = 0; i< n_vertices; i++)
    {
        free(adjList2D[i]);
        if(i>0)
        {
            sizeAdj[i] += sizeAdj[i - 1];
        }
    }

    
    
    free(adjList2D);
    // memset(LCMSize, 0, n_vertices*sizeof(int));
    //printf("%d-%d\n", totalSize, sizeAdj[n_vertices-1]);
    //printf("Got Adj List...\n Allocating on gpu mem...");

    //kernel execution time crap
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//
    checkCudaError(hipMalloc((void**)&d_adjList, totalSize * sizeof(int)), "Malloc Error d_adjList");
    checkCudaError(hipMalloc((void**)&d_sizeAdj, n_vertices * sizeof(int)), "Malloc Error d_sizeAdj");
    checkCudaError(hipMalloc((void**)&d_LCMSize, n_vertices * sizeof(int)), "Malloc Error d_sizeAdj");

    //
    hipMemcpy(d_adjList, adjList, totalSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sizeAdj, sizeAdj, n_vertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_LCMSize, LCMSize_Calc, n_vertices * sizeof(int), hipMemcpyHostToDevice);

    dim3 DimGrid(ceil((float)n_vertices/numThreads), 1, 1);
    // if (n_vertices%numThreads) 
    // {
    //     DimGrid.x++;
    // }

    dim3 DimBlock(numThreads, 1, 1);
    int totLCMSize = 0;
    //printf("Launching Size Kernel...\n");
    OPT_4_SIZES<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_LCMSize, n_vertices);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Checking Last Error, Size Kernel Launch");
    hipMemcpy(LCMSize_Calc, d_LCMSize, n_vertices * sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i = 0; i<n_vertices; i++)
    {
        totLCMSize += LCMSize_Calc[i];
        LCMSize[i] = LCMSize_Calc[i];
    }

    for(int i = 1; i<n_vertices; i++)
    {
        if(i>0)
            LCMSize[i] += LCMSize[i - 1];
    }
    //printf("%d - %d\n", totalSize, totLCMSize);
    
    lcmMatrix = (int *) malloc(totLCMSize * sizeof(int));
    memset(lcmMatrix, 0, totLCMSize*sizeof(int));
    checkCudaError(hipMalloc((void**)&d_lcmMatrix, totLCMSize * sizeof(int)), "Malloc Error d_lcmMatrix");
    hipMemcpy(d_lcmMatrix, lcmMatrix, totLCMSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_LCMSize, LCMSize, n_vertices * sizeof(int), hipMemcpyHostToDevice);
    //printf("Launching LCM Kernel...\n");
    
    // LCM_Kernel<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_lcmMatrix, d_LCMSize, n_vertices);
    OPT_4<<<DimGrid,DimBlock>>>(d_adjList, d_sizeAdj, d_lcmMatrix, d_LCMSize, n_vertices);
    
	hipDeviceSynchronize();
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Checking Last Error, Kernel Launch");
    //printf("Copying to CPU Memory...\n");
    checkCudaError(hipMemcpy(lcmMatrix, d_lcmMatrix, totLCMSize * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy Error d_lcmMatrix");
    // checkCudaError(hipMemcpy(LCMSize, d_LCMSize, n_vertices * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy Error LCMSize");
    
    // hipFree(d_lcmMatrix);
    // hipFree(d_LCMSize);
    hipFree(d_adjList);
    hipFree(d_sizeAdj);
    free(sizeAdj);
    free(adjList);

    //printf("Allocating Histogram...\n");
    int *histo, *d_histogram;
    histo = (int *) malloc(n_vertices * sizeof(int));
    memset(histo, 0, sizeof(int)*n_vertices);
    checkCudaError(hipMalloc((void**)&d_histogram, n_vertices * sizeof(int)), "Malloc Error d_histogram");
    hipMemcpy(d_histogram, histo, n_vertices * sizeof(int), hipMemcpyHostToDevice);

    //printf("Sorting LCM...\n");
    
	for(int i = 0; i< n_vertices; i++)
	{
		int offset = 0;
		if(i > 0)
		{
			offset = LCMSize[i - 1];
		}
		qsort(lcmMatrix + offset, LCMSize_Calc[i], sizeof(int), compare);
	}

    hipMemcpy(d_lcmMatrix, lcmMatrix, totLCMSize * sizeof(int), hipMemcpyHostToDevice);

	//printf("Computing Histogram...\n");
    //printf("Launching Histogram Kernel...\n");
    
    OPT_4_HIST<<<DimGrid,DimBlock>>>(d_lcmMatrix, d_LCMSize, d_histogram, n_vertices);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    checkCudaError(hipGetLastError(), "Checking Last Error, Kernel Launch");
    //printf("Copying to CPU Memory...\n");
    checkCudaError(hipMemcpy(histo, d_histogram, n_vertices * sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy Error d_lcmMatrix");

    //kernel execution stop
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
    
    printf("\nGPU OPT_4 HISTOGRAM\n");
    for(int i = 1; i < n_vertices; i++) {
        if ((long) (histo[i] / i) > 0)
            printf("%d    %d\n", i, (int) (histo[i] / i));
    }

    //kernels total times
	//printf("\n******** Processed %d Node Graph In %0.5f ms *******\n", n_vertices, elapsedTime);
	printf("******** OPT_4 Processed %d Node Graph In %0.5f sec *******\n", n_vertices, elapsedTime/1000);

	//frees stuff
    free(lcmMatrix);
    free(LCMSize_Calc);
    free(LCMSize);
    free(histo);
    hipFree(d_histogram);
    hipFree(d_lcmMatrix);
    hipFree(d_LCMSize);
	return 0;
}

__global__ void OPT_4_SIZES(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;     
    if(i<n_vertices)
    {
        int indexUsed = 0;
        int iStart = 0, iEnd = 0;
        int k = 0;

        if(i > 0)
        {       
            k = d_sizeAdj[i-1];
        }

        iEnd = d_sizeAdj[i];

        __syncthreads();

        for(int j = 0; j < n_vertices; j++) {
            if(i==j)
                continue;
            iStart = k;
            int jStart = 0, jEnd = 0;

            if(j > 0)
                jStart = d_sizeAdj[j-1];
            jEnd = d_sizeAdj[j];
            
            int compVec = 0;

            while (iStart < iEnd && jStart < jEnd)
            {
                    if(d_adjList[iStart] < d_adjList[jStart])
                        iStart++;
                    else if (d_adjList[jStart] < d_adjList[iStart])
                        jStart++;
                    else // if arr1[i] == arr2[j] 
                    {
                        jStart++;
                        iStart++;
                        compVec++;
                        break;
                    }
            }

            if (compVec > 0)
            {
                indexUsed++;
            }
        }
    
        __syncthreads();
        d_LCMSize[i] = indexUsed;
        // __syncthreads();
    
    }

}

__global__ void OPT_4(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;     
    if(i<n_vertices)
    {
        int indexUsed = 0, indexOffset = 0;
        int iStart = 0, iEnd = 0;
        int k = 0;

        if(i > 0)
        {       
            k = d_sizeAdj[i-1];
            indexOffset = d_LCMSize[i-1];
        }

        iEnd = d_sizeAdj[i];
        
        for(int j = indexOffset; j<iEnd; j++)
        {
            d_lcmMatrix[j] = 0;
        }

        __syncthreads();

        for(int j = 0; j < n_vertices; j++) {
            if(i==j)
                continue;
            iStart = k;
            int jStart = 0, jEnd = 0;

            if(j > 0)
                jStart = d_sizeAdj[j-1];
            jEnd = d_sizeAdj[j];
            
            int compVec = 0;

            while (iStart < iEnd && jStart < jEnd)
            {
                    if(d_adjList[iStart] < d_adjList[jStart])
                        iStart++;
                    else if (d_adjList[jStart] < d_adjList[iStart])
                        jStart++;
                    else // if arr1[i] == arr2[j] 
                    {
                        jStart++;
                        iStart++;
                        compVec++;
                    }
            }

            if (compVec > 0)
            {
                atomicAdd((int*)&d_lcmMatrix[indexUsed + indexOffset], compVec);
                // d_lcmMatrix[indexUsed + indexOffset] = compVec;
                indexUsed++;
            }
            // __syncthreads();
        }
    
        // __syncthreads();
        // d_LCMSize[i] = indexUsed;
        // __syncthreads();
    
    }

}

__global__ void OPT_4_HIST(int *d_lcmMatrix, int *d_LCMSize, int *d_histogram, int n_vertices)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int count = 0, countMax = -1;
  
    if(i<n_vertices)
    {
        int iStart = 0;
        if(i>0)
            iStart = d_LCMSize[i - 1]; //Offset
        count = 0;
        int iSize = d_LCMSize[i] - iStart;

        for(int j = 0; j < n_vertices; j++) {
            int jStart = 0;
            if(j>0)
                jStart = d_LCMSize[j - 1]; //Offset

            int jSize = d_LCMSize[j] - jStart;
            if(iSize != jSize)
                continue;
      
            int eq = 1;
            for(int k = 0; k < iSize; k++)
            {
                if(d_lcmMatrix[iStart + k] != d_lcmMatrix[jStart + k])
                {
                    eq = 0;
                    break;
                }
            }
            if(eq == 1)
            {               
                count++;
            }
        }

        if(countMax < count)
            countMax = count;
        atomicAdd((int*)&d_histogram[count], 1);
        // d_histogram[count]++;
    }
}

//OPTIMIZED CPU CODE
void LCM_CPU(igraph_t &graph, igraph_neimode_t OUTALL)
{
	int n_vertices = igraph_vcount(&graph);
	igraph_adjlist_t al;
	igraph_adjlist_init(&graph, &al, OUTALL);
	igraph_adjlist_simplify(&al);

	long int **adjList;
	int *sizeAdj;
	igraph_vector_int_t *adjVec;

	adjList = (long int **) calloc(n_vertices, sizeof(long int *));
	sizeAdj = (int *) calloc(n_vertices, sizeof(int));
	for (int i = 0; i < n_vertices; i++) {
		adjVec = igraph_adjlist_get(&al, i);

		adjList[i] = (long int *) calloc(igraph_vector_int_size(adjVec), sizeof(long int));
		sizeAdj[i] = (int) igraph_vector_int_size(adjVec);
		for(int k = 0; k< igraph_vector_int_size(adjVec); k++)
		{
			adjList[i][k] = (long int) VECTOR(*adjVec)[k];
		}
	}

	for(int i = 0; i< n_vertices; i++)
	{
		qsort(adjList[i], sizeAdj[i], sizeof(long int), compare);
	}

	LCM_CPU_Kernel(adjList, sizeAdj, n_vertices);
}

void LCM_CPU_Kernel(long int **adjList, int *sizeAdj, int n_vertices)
{
	Array *lcmMatrix;
	lcmMatrix = (Array *) calloc(n_vertices, sizeof(Array));
	for(int i = 0; i < n_vertices; i++) {
		initArray(&lcmMatrix[i], sizeAdj[i]);
	}
	//finds similar vertices
	for(int i = 0; i < n_vertices; i++) {
		
		long int* neisVec1 = adjList[i];
		//inner loop
		for(int j = i+1; j < n_vertices; j++) {
			long int* neisVec2 = adjList[j];
			int compVec = commonNeighbor(neisVec1, neisVec2, sizeAdj[i], sizeAdj[j]);
			if (compVec > 0)
			{
				insertArray(&lcmMatrix[i], compVec);
				insertArray(&lcmMatrix[j], compVec);
			}
		}
	}
	//printf("Finished Computing LCM\n");
	for(int i = 0; i < n_vertices; i++) {
		qsort(lcmMatrix[i].array, lcmMatrix[i].used, sizeof(int), compare);
		// printf("%d:\t", i);
		// for(int j=0;j < lcmMatrix[i].used; j++)
		// {
		// 	printf("%d-", lcmMatrix[i].array[j]);
		// }
		// printf("\n");
	}
	
	long int histo[n_vertices];
	memset(histo, 0, sizeof(long int)*n_vertices);
	int count = 0, countMax = -1;

	for(int i = 0; i < n_vertices; i++) {
		count = 0;
		for(int j = 0; j < n_vertices; j++) {
			if(lcmMatrix[i].used != lcmMatrix[j].used)
				continue;
			int eq = equalArray(lcmMatrix[i],lcmMatrix[j]);
			if(eq == 1)
			{				
				count++;
			}
		}

		if(countMax < count)
			countMax = count;
		histo[count]++;
	}

	printf("\nCPU OPTIMIZED HISTOGRAM\n");
	for(int i = 1; i <= countMax; i++) {
		if ((long) (histo[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (histo[i] / i));
	}

}

void initArray(Array *a, size_t initialSize) {
  a->array = (int *)malloc(initialSize * sizeof(int));
  a->used = 0;
  a->size = initialSize;
}

void insertArray(Array *a, int element) {
  // a->used is the number of used entries, because a->array[a->used++] updates a->used only *after* the array has been accessed.
  // Therefore a->used can go up to a->size 
  if (a->used == a->size) {
    a->size *= 2;
    a->array = (int *)realloc(a->array, a->size * sizeof(int));
  }
  a->array[a->used++] = element;
}

void freeArray(Array *a) {
  free(a->array);
  a->array = NULL;
  a->used = a->size = 0;
}

int commonNeighbor(long int arr1[], long int arr2[], int m, int n)
{
  int i = 0, j = 0;
  int numCommon = 0;
  while (i < m && j < n)
  {
    if (arr1[i] < arr2[j])
      i++;
    else if (arr2[j] < arr1[i])
      j++;
    else /* if arr1[i] == arr2[j] */
    {
      // printf(" %d ", arr2[j++]);
      j++;
      i++;
      numCommon++;
    }
  }
  return numCommon;
}

int equalArray(Array a1, Array a2)
{
	if( a1.used != a2.used)
	{
		return 0;
	}
	for(int i = 0; i < a1.used; i++)
	{
		if(a1.array[i] != a2.array[i])
			return 0;
	}
	return 1;

}

int compare(const void* a, const void* b) {
	return ( *(int*)a - *(int*)b );
}

//naive cpu version, slow and takes a shit load of host memory
//uses adjacency matrix on cpu
void LCM_cpu_baseline(igraph_t &graph) {

	//gets num vertices and allocates, sets to zero adj matrix
	int n_vertices = igraph_vcount(&graph), vsize;
	int *matrix = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(matrix, 0, sizeof(int)*n_vertices*n_vertices);

	//graph vector and initializes it to zero
	igraph_vector_t vec;
	igraph_vector_init(&vec, 0);

	//builds adj matrix
	for(int i = 0; i < n_vertices; i++) {

		//grabs neighbors and size
		igraph_neighbors(&graph, &vec, i, OUTALL);
		vsize = igraph_vector_size(&vec);

		//adds ones where its adjacent
		for(int j = 0; j < vsize; j++) {

			matrix[i*n_vertices + (int)VECTOR(vec)[j]] = 1;
		}
	}

	//result adj matrix set to zero
	int *result = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	memset(result, 0, sizeof(int)*n_vertices*n_vertices);
	int cval;

	//multiplies it against itself
	for(int i = 0; i < n_vertices; i++) {

		for(int j = i+1; j < n_vertices; j++) {

			cval = 0;

			for(int k = 0; k < n_vertices; k++)
				cval += matrix[i*n_vertices + k] * matrix[k*n_vertices + j];

			//enters val and transposes
			result[i*n_vertices + j] = cval;
			result[j*n_vertices + i] = cval;
		}

		//sorts the vertice/row
		qsort(&result[i*n_vertices], n_vertices + 0, sizeof(int), compare);
	}

	//multiplies it against itself, REALL SLOW CODE LOL
	// int *result = (int *)malloc(n_vertices*n_vertices*sizeof(int));
	// memset(result, 0, sizeof(int)*n_vertices*n_vertices);
	// int cval;

	// for(int i = 0; i < n_vertices; i++) {

	// 	for(int j = 0; j < n_vertices; j++) {

	// 		cval = 0;

	// 		for(int k = 0; k < n_vertices; k++) {

	// 			cval += matrix[i*n_vertices + k] * matrix[k*n_vertices + j];
	// 		}

	// 		result[i*n_vertices + j] = cval;
	// 	}

	// 	qsort(&result[i*n_vertices], n_vertices +1, sizeof(int), compare);
	// }

	//histogram
	bool equal;
	int count, countMax = -1;
	int *hist = (int*)malloc(sizeof(int) * n_vertices);
	memset(hist, 0, sizeof(int)*n_vertices);

	for(int i = 0; i < n_vertices; i++) {

		count = 0;

		for(int j = 0; j < n_vertices; j++) {

			equal = false;

			for(int k = 0; k < n_vertices; k++) {

				if(result[i*n_vertices + k] == result[j*n_vertices + k])
					equal = true;
				else {
					equal = false;
					break;
				}
			}

			if(equal)
				++count;
		}
		if(countMax < count)
				countMax = count;

		++hist[count];
	}

	//prints results
	printf("\nCPU NAIVE HISTOGRAM\n");
	for(int i = 1; i <= countMax; i++) {
		if ((long) (hist[i] / i) > 0)
			printf("%d    %ld\n", i, (long) (hist[i] / i));
	}

	//frees shit
	free(matrix);
	free(result);
	free(hist);
}